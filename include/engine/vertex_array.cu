#include "assert.h"
#include "engine/vertex_array.hpp"
#include "hip/hip_runtime.h"
VertexArray::Iterator VertexArray::begin()
{
    return VertexArray::Iterator(*this);
}
VertexArray::Iterator VertexArray::end()
{
    return VertexArray::Iterator(*this) + this->size();
}

VertexArray::Iterator VertexArray::Iterator::operator++(int)
{
    Iterator temp = *this;
    ++current;
    return temp;
}
VertexArray::Iterator &VertexArray::Iterator::operator++()
{
    ++current;
    return *this;
}

VertexArray::Iterator VertexArray::Iterator::operator+(int x) const
{
    Iterator temp = *this;
    temp.advance(x);
    return temp;
}

void VertexArray::Iterator::advance(int n)
{
    current += n;
}

VertexArray::Iterator::Iterator(VertexArray &va) : current(va.vertices) {}
Vertex &VertexArray::Iterator::operator*() const { return *current; }
Vertex *VertexArray::Iterator::operator->() const { return current; }

bool VertexArray::Iterator::operator==(const Iterator &other) const
{
    return current == other.current;
}
bool VertexArray::Iterator::operator!=(const Iterator &other) const
{
    return current != other.current;
}

VertexArray::VertexArray()
    : vertices(nullptr),
      max_size(0),
      current_size(0) {}
VertexArray::VertexArray(const size_t max_size = 3)
    : max_size(max_size),
      current_size(0)
{
    assert(max_size > 0 && "Max size must be positive");
    if (hipMallocManaged<Vertex>(&vertices, max_size, hipMemAttachHost) != hipSuccess)
    {
        throw std::runtime_error("Failed to allocate memory while copying vertex array");
    }
}
VertexArray::VertexArray(VertexArray &&va)
    : max_size(va.max_size),
      current_size(va.current_size),
      vertices(va.vertices)
{
    va.vertices = nullptr;
}
VertexArray::VertexArray(const VertexArray &other)
    : max_size(other.max_size),
      current_size(other.current_size)
{
    if (hipMallocManaged<Vertex>(&vertices, max_size, hipMemAttachHost) != hipSuccess)
    {
        throw std::runtime_error("Failed to allocate memory while copying vertex array");
    }
    if (hipMemcpy(vertices, other.vertices, sizeof(Vertex) * current_size, hipMemcpyDeviceToDevice) != hipSuccess)
    {
        throw std::runtime_error("Failed to copy vertex array");
    }
}
VertexArray::~VertexArray()
{
    if (vertices != nullptr)
    {
        hipFree(vertices);
        vertices = nullptr;
    }
}

VertexArray &VertexArray::operator=(const VertexArray &other)
{
    if (vertices != nullptr)
        hipFree(vertices);

    max_size = other.max_size;
    current_size = other.current_size;
    if (hipMallocManaged<Vertex>(&vertices, max_size, hipMemAttachHost) != hipSuccess)
    {
        throw std::runtime_error("Failed to allocate memory while copying vertex array");
    }
    if (hipMemcpy(vertices, other.vertices, sizeof(Vertex) * current_size, hipMemcpyDeviceToDevice) != hipSuccess)
    {
        throw std::runtime_error("Failed to copy vertex array");
    }
}
VertexArray &VertexArray::operator=(VertexArray &&other)
{
    if (vertices != nullptr)
        hipFree(vertices);
    vertices = other.vertices;
    other.vertices = nullptr;
    max_size = other.max_size;
    current_size = other.current_size;
}

void VertexArray::push_back(const Vertex &v)
{
    current_size++;
    if (current_size > max_size)
    {
        Vertex *old = vertices;
        if (hipMallocManaged<Vertex>(&vertices, current_size, hipMemAttachHost) != hipSuccess)
        {
            throw std::runtime_error("Can't alloc memory for push back Vertex");
        }
        if (hipMemcpy(vertices, old, max_size, hipMemcpyHostToHost) != hipSuccess)
        {
            throw std::runtime_error("Can't copy memory for push back Vertex");
        }
    }
    else
    {
        vertices[current_size-1] = v;//copy
    }
}

void VertexArray::push_back(Vertex &&v)
{
}
void VertexArray::pop_back()
{
}
// Remove an vertex from the array, with specified index
void VertexArray::pop(int index)
{
}